#include "hip/hip_runtime.h"
/******************************************************************************
# SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
# SPDX-License-Identifier: Apache-2.0
#
# Licensed under the Apache License, Version 2.0 (the "License");
# you may not use this file except in compliance with the License.
# You may obtain a copy of the License at
#
# http://www.apache.org/licenses/LICENSE-2.0
#
# Unless required by applicable law or agreed to in writing, software
# distributed under the License is distributed on an "AS IS" BASIS,
# WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
# See the License for the specific language governing permissions and
# limitations under the License.
******************************************************************************/

#include "check.h"
#include "optimizer.h"
#include "optimizer_kernel.cuh"
#include "torch_utils.h"
#include "utils.h"
#include "index_calculation.h"
#include <iostream>

void find_pointers(
  std::shared_ptr<dyn_emb::DynamicVariableBase> table,
  const size_t n,
  const at::Tensor keys,
  at::Tensor values,
  at::Tensor founds);

void find_or_insert_pointers(
  std::shared_ptr<dyn_emb::DynamicVariableBase> table,
  const size_t n,
  const at::Tensor keys,
  at::Tensor values,
  at::Tensor founds,
  const std::optional<uint64_t> score = std::nullopt,
  bool unique_key = true,
  bool ignore_evict_strategy = false);

void insert_and_evict(
    std::shared_ptr<dyn_emb::DynamicVariableBase> table,
    const size_t n,
    const at::Tensor keys,
    const at::Tensor values,
    const std::optional<uint64_t> score,
    at::Tensor evicted_keys,
    at::Tensor evicted_values,
    at::Tensor evicted_score,
    at::Tensor d_evicted_counter,
    bool unique_key = true,
    bool ignore_evict_strategy = false);

int64_t find_and_get_missed(
  std::shared_ptr<dyn_emb::DynamicVariableBase> ht,
  uint64_t n,
  at::Tensor keys,
  at::Tensor foundss,
  at::Tensor vals_ptr,
  at::Tensor missed_keys,
  at::Tensor missed_ids,
  at::Tensor reverse_ids
);

at::Tensor create_sub_tensor(const at::Tensor &original_tensor,
                             int64_t offset);

void insert_or_assign(std::shared_ptr<dyn_emb::DynamicVariableBase> table,
                      const size_t n, const at::Tensor keys,
                      const at::Tensor values,
                      const c10::optional<at::Tensor> &score = c10::nullopt,
                      bool unique_key = true,
                      bool ignore_evict_strategy = false);

void assign(std::shared_ptr<dyn_emb::DynamicVariableBase> table, const size_t n,
            const at::Tensor keys, const at::Tensor values,
            const c10::optional<at::Tensor> &score = c10::nullopt,
            bool unique_key = true);

namespace dyn_emb {

constexpr int MULTIPLIER = 4;
constexpr int WARPSIZE = 32;
constexpr int OPTIMIZER_BLOCKSIZE_VEC = 64;
constexpr int OPTIMIZER_BLOCKSIZE = 1024;

template<typename IdxType, typename V, int GROUP_SIZE=32>
__global__ void get_missing_values(
    int n, int dim,
    IdxType const * __restrict__ original_ids,
    V const* __restrict__ src, 
    V* __restrict__ dst) {
  
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int group_id = tid / GROUP_SIZE;
  int lane_id = tid % GROUP_SIZE;
  if (group_id < n) {
    auto src_id = original_ids[group_id];
    for (int i = lane_id; i < dim; i += GROUP_SIZE) {
      dst[group_id * dim + i] = src[src_id * dim + i];
    }
  }
}

void update_heirarchical_tables(
  std::shared_ptr<dyn_emb::DynamicVariableBase> t1,
  std::shared_ptr<dyn_emb::DynamicVariableBase> t2,
  int64_t total_key_num,
  int64_t found_key_num,
  at::Tensor keys,
  at::Tensor values,
  const std::optional<uint64_t> score
) {

  //1.update
  if (t1->need_score()) {
    if (not score) {
      throw std::invalid_argument("Must specify the score.");
    }
    auto&& option = at::TensorOptions().dtype(at::kUInt64).device(keys.device());
    // broadcast scores
    at::Tensor bc_scores = at::empty({static_cast<int64_t>(found_key_num)}, option);
    bc_scores.fill_(score.value());
    c10::optional<at::Tensor> opt_scores(bc_scores);
    assign(t1, found_key_num, keys, values, opt_scores);
  } else {
    assign(t1, found_key_num, keys, values);
  }

  if (found_key_num == total_key_num) return;
  auto stream = at::cuda::getCurrentCUDAStream().stream();

  // 2.insert_and_evict
  int64_t missed_key_num = total_key_num - found_key_num;
  auto missed_keys = create_sub_tensor(keys, found_key_num);
  auto missed_values = create_sub_tensor(values, found_key_num * values.size(1));
  at::Tensor evicted_keys = at::empty({static_cast<int64_t>(missed_key_num)}, keys.options());
  at::Tensor evicted_values = at::empty({static_cast<int64_t>(missed_key_num), values.size(1)}, values.options());
  at::Tensor evicted_score = at::empty({static_cast<int64_t>(missed_key_num)}, keys.options().dtype(at::kUInt64));
  at::Tensor d_evicted_counter =  at::zeros({static_cast<int64_t>(1)}, at::TensorOptions().dtype(at::kUInt64).device(keys.device()));
  insert_and_evict(t1, missed_key_num, missed_keys, missed_values, score, evicted_keys, evicted_values, evicted_score, d_evicted_counter);
  uint64_t evict_counter = 0;
  AT_CUDA_CHECK(hipMemcpyAsync(&evict_counter, d_evicted_counter.data_ptr(),
      sizeof(uint64_t), hipMemcpyDeviceToHost, stream));
  AT_CUDA_CHECK(hipStreamSynchronize(stream));

  if (evict_counter > missed_key_num) {
    throw std::runtime_error("Evict too much keys than new inserted.");
  }
  
  auto evict_score_opt = c10::make_optional(evicted_score);
  insert_or_assign(t2, evict_counter, evicted_keys, evicted_values, evict_score_opt);
}

__global__ void get_found_counter(bool const * __restrict__ found, int64_t* counter, int64_t batch_size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < batch_size) {
    bool cur_found = found[tid];
    bool pre_found = tid != 0 ? found[tid - 1] : true;
    if (cur_found != pre_found) {
      *counter = static_cast<int64_t>(tid);
    }
    if ((tid == (batch_size - 1)) && cur_found) {
      *counter = batch_size;
    }
  }
}

__global__ void verify_found_counter(bool const * __restrict__ found, int64_t counter, int64_t batch_size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < batch_size) {
    bool found_ = found[tid];
    if (tid < counter) {
      if (not found_) {
        asm("trap;");
      }
    } else {
      if (found_) {
        asm("trap;");
      }  
    }
  }
}

template <typename T>
__global__ void set_uncached_pointer_kernel(
  int num_uncached,
  T* const * __restrict__ in_values_ptr,
  int const * __restrict__ outs_idx,
  bool* __restrict__ out_founds,
  T* * __restrict__ out_values_ptr
) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num_uncached) {
    T* const in_value_ptr = in_values_ptr[tid];
    int const out_idx = outs_idx[tid];
    if (in_value_ptr != nullptr) {
      out_founds[out_idx] = true;
      out_values_ptr[out_idx] = in_value_ptr;
    }
  }
}

int64_t find_ptr_from_hierarchical_table_for_classified_keys(
  std::shared_ptr<dyn_emb::DynamicVariableBase> ht1,
  std::shared_ptr<dyn_emb::DynamicVariableBase> ht2,
  uint64_t n,
  at::Tensor keys,
  at::Tensor founds,
  at::Tensor vals_ptr
) {

  find_pointers(ht1, n, keys, vals_ptr, founds);
  auto found_counter = at::empty({static_cast<int64_t>(1)},
    at::TensorOptions().dtype(at::kLong).device(keys.device()));
  
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  
  get_found_counter<<<(n + 127) / 128, 128, 0, stream>>>(
    founds.data_ptr<bool>(), found_counter.data_ptr<int64_t>(), n
  );
  DEMB_CUDA_KERNEL_LAUNCH_CHECK();
  int64_t found_counter_host = 0;
  AT_CUDA_CHECK(hipMemcpyAsync(&found_counter_host, found_counter.data_ptr(),
      sizeof(int64_t), hipMemcpyDeviceToHost, stream));
  AT_CUDA_CHECK(hipStreamSynchronize(stream));
  ///TODO: open it under debug mode
  // verify_found_counter<<<(n + 127) / 128, 128, 0, stream>>>(
  //   founds.data_ptr<bool>(), found_counter_host, n
  // );
  // DEMB_CUDA_KERNEL_LAUNCH_CHECK();
  // AT_CUDA_CHECK(hipStreamSynchronize(stream));
  if (found_counter_host == n) {
    return found_counter_host;
  }
  auto missed_keys = create_sub_tensor(keys, found_counter_host);
  auto vals_host_ptr = create_sub_tensor(vals_ptr, found_counter_host);
  auto founds_host = create_sub_tensor(founds, found_counter_host);
  int64_t missed_counter = n - found_counter_host;
  find_pointers(ht2, missed_counter, missed_keys, vals_host_ptr, founds_host);
  return found_counter_host;
}

void storage_find_pointers(
  std::shared_ptr<dyn_emb::DynamicVariableBase> storage,
  int64_t n,
  at::Tensor const& indices,
  at::Tensor& founds, // cache
  at::Tensor& weight_ptrs
) {
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  auto key_type = storage->key_type();
  at::Tensor inv_founds = at::logical_not(founds);
  at::Tensor num_missing = at::empty({static_cast<int64_t>(2)}, indices.options().dtype(at::kInt));
  at::Tensor missing_keys = at::empty_like(indices);
  DISPATCH_INTEGER_DATATYPE_FUNCTION(key_type, KeyType, [&] {
    select_async<KeyType>(n, inv_founds.data_ptr<bool>(), reinterpret_cast<KeyType*>(indices.data_ptr()),
      reinterpret_cast<KeyType*>(missing_keys.data_ptr()), num_missing.data_ptr<int>(), indices.device(), stream);
  });
  at::Tensor missing_keys_idx = at::empty({static_cast<int64_t>(n)}, indices.options().dtype(at::kInt));
  select_index_async(n, inv_founds.data_ptr<bool>(), missing_keys_idx.data_ptr<int>(), 
                    num_missing.data_ptr<int>()+1, indices.device(), stream);
  
  int h_num_missing = 0;
  AT_CUDA_CHECK(hipMemcpyAsync(&h_num_missing, num_missing.data_ptr<int>(), sizeof(int), hipMemcpyDeviceToHost, stream));
  AT_CUDA_CHECK(hipStreamSynchronize(stream));
  if (h_num_missing != 0) {
    auto vals_host_ptr = at::empty({static_cast<int64_t>(h_num_missing)}, missing_keys.options().dtype(at::kLong));
    auto founds_host = at::empty({static_cast<int64_t>(h_num_missing)}, missing_keys.options().dtype(at::kBool));
    find_pointers(storage, h_num_missing, missing_keys, vals_host_ptr, founds_host);
    auto value_type = storage->value_type();
    DISPATCH_FLOAT_DATATYPE_FUNCTION(value_type, ValueType, [&] {
      set_uncached_pointer_kernel<ValueType><<<(h_num_missing + 63) / 64, 64, 0, stream>>>(
        h_num_missing, reinterpret_cast<ValueType**>(vals_host_ptr.data_ptr()), 
        missing_keys_idx.data_ptr<int>(), founds.data_ptr<bool>(),
        reinterpret_cast<ValueType**>(weight_ptrs.data_ptr())
      );
    });
    DEMB_CUDA_KERNEL_LAUNCH_CHECK();
  }
}

void dynamic_emb_sgd_with_table(
    std::shared_ptr<dyn_emb::DynamicVariableBase> table, const uint64_t n, 
    const at::Tensor indices, const at::Tensor grads, const float lr, DataType weight_type, 
    const std::optional<uint64_t> score, const c10::optional<at::Tensor>& embs,
    const std::optional<std::shared_ptr<dyn_emb::DynamicVariableBase>> host_table) {

  if (n == 0) return;
  TORCH_CHECK(indices.is_cuda(), "indices must be a CUDA tensor");
  TORCH_CHECK(grads.is_cuda(), "grads must be a CUDA tensor");

  at::Tensor founds = at::empty({static_cast<int64_t>(n)}, 
                                at::TensorOptions().dtype(at::kBool).device(indices.device()));
  at::Tensor weight_ptrs = at::empty({static_cast<int64_t>(n)}, 
                                     at::TensorOptions().dtype(at::kLong).device(indices.device()));

  auto &device_prop = DeviceProp::getDeviceProp(grads.device().index());

  int64_t dim = grads.size(1);
  int64_t ev_nums = weight_ptrs.size(0);

  auto grad_type =
      scalartype_to_datatype(convertTypeMetaToScalarType(grads.dtype()));

  auto stream = at::cuda::getCurrentCUDAStream().stream();
  find_pointers(table, n, indices, weight_ptrs, founds);

  if (host_table.has_value()) {
    storage_find_pointers(host_table.value(), n, indices, founds, weight_ptrs);
  }

  DISPATCH_FLOAT_DATATYPE_FUNCTION(grad_type, g_t, [&] {
    DISPATCH_FLOAT_DATATYPE_FUNCTION(weight_type, w_t, [&] {
      
      SgdVecOptimizer<g_t, w_t> opt{lr};
      if (dim % 4 == 0) {
        const int max_grid_size =
            device_prop.num_sms *
            (device_prop.max_thread_per_sm / OPTIMIZER_BLOCKSIZE_VEC);
        const int warp_per_block = OPTIMIZER_BLOCKSIZE_VEC / WARPSIZE;

        int grid_size = 0;
        if (ev_nums / warp_per_block < max_grid_size) {
          grid_size = (ev_nums - 1) / warp_per_block + 1;
        } else if (ev_nums / warp_per_block > max_grid_size * MULTIPLIER) {
          grid_size = max_grid_size * MULTIPLIER;
        } else {
          grid_size = max_grid_size;
        }

        auto kernel = update4_kernel<g_t, w_t, decltype(opt)>;
        kernel<<<grid_size, OPTIMIZER_BLOCKSIZE_VEC, 0, stream>>>(
          ev_nums, dim, reinterpret_cast<const g_t *>(grads.data_ptr()),
          reinterpret_cast<w_t **>(weight_ptrs.data_ptr()), founds.data_ptr<bool>(), opt);
        DEMB_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        int block_size = dim > OPTIMIZER_BLOCKSIZE ? OPTIMIZER_BLOCKSIZE : dim;
        int grid_size = ev_nums;

        auto kernel = update_kernel<g_t, w_t, decltype(opt)>;
        kernel<<<grid_size, block_size, 0, stream>>>(
          ev_nums, dim, reinterpret_cast<const g_t *>(grads.data_ptr()),
          reinterpret_cast<w_t **>(weight_ptrs.data_ptr()), founds.data_ptr<bool>(), opt);
        DEMB_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  });
  DEMB_CUDA_KERNEL_LAUNCH_CHECK();
}

void dynamic_emb_adam_with_table(
  std::shared_ptr<dyn_emb::DynamicVariableBase> ht,
  const uint64_t n, const at::Tensor indices, const at::Tensor grads, 
  const float lr, const float beta1, const float beta2, const float eps,
  const float weight_decay,
  const uint32_t iter_num, DataType weight_type, 
  const std::optional<uint64_t> score,
  const c10::optional<at::Tensor>& embs,
  const std::optional<std::shared_ptr<dyn_emb::DynamicVariableBase>> host_table) {

  if (n == 0) return;
  TORCH_CHECK(indices.is_cuda(), "indices must be a CUDA tensor");
  TORCH_CHECK(grads.is_cuda(), "grads must be a CUDA tensor");

  at::Tensor founds = at::empty({static_cast<int64_t>(n)}, 
                                at::TensorOptions().dtype(at::kBool).device(indices.device()));
  at::Tensor vector_ptrs = at::empty({static_cast<int64_t>(n)}, 
                                     at::TensorOptions().dtype(at::kLong).device(indices.device()));

  auto &device_prop = DeviceProp::getDeviceProp(grads.device().index());

  int64_t dim = grads.size(1);
  int64_t ev_nums = n;

  auto grad_type =
      scalartype_to_datatype(convertTypeMetaToScalarType(grads.dtype()));

  auto stream = at::cuda::getCurrentCUDAStream().stream();

  find_pointers(ht, n, indices, vector_ptrs, founds);

  if (host_table.has_value()) {
    storage_find_pointers(host_table.value(), n, indices, founds, vector_ptrs);
  }

  DISPATCH_FLOAT_DATATYPE_FUNCTION(grad_type, g_t, [&] {
    DISPATCH_FLOAT_DATATYPE_FUNCTION(weight_type, w_t, [&] {
      AdamVecOptimizer<g_t, w_t> opt{lr,
                                     beta1,
                                     beta2,
                                     eps,
                                     weight_decay,
                                     iter_num};
      if (dim % 4 == 0) {
        const int max_grid_size =
            device_prop.num_sms *
            (device_prop.max_thread_per_sm / OPTIMIZER_BLOCKSIZE_VEC);
        const int warp_per_block = OPTIMIZER_BLOCKSIZE_VEC / WARPSIZE;

        int grid_size = 0;
        if (ev_nums / warp_per_block < max_grid_size) {
          grid_size = (ev_nums - 1) / warp_per_block + 1;
        } else if (ev_nums / warp_per_block > max_grid_size * MULTIPLIER) {
          grid_size = max_grid_size * MULTIPLIER;
        } else {
          grid_size = max_grid_size;
        }

        auto kernel = update4_kernel<g_t, w_t, decltype(opt)>;
        kernel<<<grid_size, OPTIMIZER_BLOCKSIZE_VEC, 0, stream>>>(
          ev_nums, dim, reinterpret_cast<const g_t *>(grads.data_ptr()),
          reinterpret_cast<w_t **>(vector_ptrs.data_ptr()), founds.data_ptr<bool>(), opt);
        DEMB_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        int block_size = dim > OPTIMIZER_BLOCKSIZE ? OPTIMIZER_BLOCKSIZE : dim;
        int grid_size = ev_nums;

        auto kernel = update_kernel<g_t, w_t, decltype(opt)>;
        kernel<<<grid_size, block_size, 0, stream>>>(
          ev_nums, dim, reinterpret_cast<const g_t *>(grads.data_ptr()),
          reinterpret_cast<w_t **>(vector_ptrs.data_ptr()), founds.data_ptr<bool>(), opt);
        DEMB_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  });
  DEMB_CUDA_KERNEL_LAUNCH_CHECK();
}

void dynamic_emb_adagrad_with_table(
  std::shared_ptr<dyn_emb::DynamicVariableBase> ht,
  const uint64_t n, const at::Tensor indices,
  const at::Tensor grads,
  const float lr,
  const float eps,
  DataType weight_type,const std::optional<uint64_t> score,
  const c10::optional<at::Tensor>& embs,
  const std::optional<std::shared_ptr<dyn_emb::DynamicVariableBase>> host_table){
  if (n == 0) return;

  TORCH_CHECK(indices.is_cuda(), "indices must be a CUDA tensor");
  TORCH_CHECK(grads.is_cuda(), "grads must be a CUDA tensor");

  at::Tensor founds = at::empty({static_cast<int64_t>(n)}, 
                                at::TensorOptions().dtype(at::kBool).device(indices.device()));
  at::Tensor vector_ptrs = at::empty({static_cast<int64_t>(n)}, 
                                     at::TensorOptions().dtype(at::kLong).device(indices.device()));

  auto& device_prop = DeviceProp::getDeviceProp(grads.device().index());

  int64_t dim = grads.size(1);
  int64_t ev_nums = n;

  auto grad_type = scalartype_to_datatype(convertTypeMetaToScalarType(grads.dtype()));

  auto stream = at::cuda::getCurrentCUDAStream().stream();

  find_pointers(ht, n, indices, vector_ptrs, founds);

  if (host_table.has_value()) {
    storage_find_pointers(host_table.value(), n, indices, founds, vector_ptrs);
  }

  DISPATCH_FLOAT_DATATYPE_FUNCTION(grad_type, g_t, [&] {
    DISPATCH_FLOAT_DATATYPE_FUNCTION(weight_type, w_t, [&] {

      AdaGradVecOptimizer<g_t,w_t> opt{lr, eps};

      if (dim % 4 == 0) {
        const int max_grid_size = device_prop.num_sms * (device_prop.max_thread_per_sm / OPTIMIZER_BLOCKSIZE_VEC);
        const int warp_per_block = OPTIMIZER_BLOCKSIZE_VEC/WARPSIZE;

        int grid_size = 0;
        if (ev_nums/warp_per_block < max_grid_size){
            grid_size = (ev_nums-1)/warp_per_block+1;
        }
        else if (ev_nums/warp_per_block > max_grid_size*MULTIPLIER){
            grid_size = max_grid_size*MULTIPLIER;
        }
        else{
            grid_size = max_grid_size;
        }

        auto kernel = update4_kernel<g_t, w_t, decltype(opt)>;
        kernel<<<grid_size, OPTIMIZER_BLOCKSIZE_VEC, 0, stream>>>(
          ev_nums, dim, reinterpret_cast<const g_t *>(grads.data_ptr()),
          reinterpret_cast<w_t **>(vector_ptrs.data_ptr()), founds.data_ptr<bool>(), opt);
        DEMB_CUDA_KERNEL_LAUNCH_CHECK();
      } else {

        int block_size = dim > OPTIMIZER_BLOCKSIZE ? OPTIMIZER_BLOCKSIZE : dim;
        int grid_size = ev_nums;

        auto kernel = update_kernel<g_t, w_t, decltype(opt)>;
        kernel<<<grid_size, block_size, 0, stream>>>(
          ev_nums, dim, reinterpret_cast<const g_t *>(grads.data_ptr()),
          reinterpret_cast<w_t **>(vector_ptrs.data_ptr()), founds.data_ptr<bool>(), opt);
        DEMB_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  });
  DEMB_CUDA_KERNEL_LAUNCH_CHECK();
}

void dynamic_emb_rowwise_adagrad_with_table(
  std::shared_ptr<dyn_emb::DynamicVariableBase> ht,
  const uint64_t n, const at::Tensor indices,
  const at::Tensor grads,
  const float lr,
  const float eps,
  DataType weight_type,const std::optional<uint64_t> score,
  const c10::optional<at::Tensor>& embs,
  const std::optional<std::shared_ptr<dyn_emb::DynamicVariableBase>> host_table) {
  if (n == 0) return;
  TORCH_CHECK(indices.is_cuda(), "indices must be a CUDA tensor");
  TORCH_CHECK(grads.is_cuda(), "grads must be a CUDA tensor");

  at::Tensor founds = at::empty({static_cast<int64_t>(n)}, 
                                at::TensorOptions().dtype(at::kBool).device(indices.device()));
  at::Tensor vector_ptrs = at::empty({static_cast<int64_t>(n)}, 
                                     at::TensorOptions().dtype(at::kLong).device(indices.device()));

  auto& device_prop = DeviceProp::getDeviceProp(grads.device().index());

  int64_t dim = grads.size(1);
  int64_t ev_nums = n;

  auto grad_type = scalartype_to_datatype(convertTypeMetaToScalarType(grads.dtype()));

  auto stream = at::cuda::getCurrentCUDAStream().stream();

  find_pointers(ht, n, indices, vector_ptrs, founds);

  if (host_table.has_value()) {
    storage_find_pointers(host_table.value(), n, indices, founds, vector_ptrs);
  }

  DISPATCH_FLOAT_DATATYPE_FUNCTION(grad_type, g_t, [&] {
    DISPATCH_FLOAT_DATATYPE_FUNCTION(weight_type, w_t, [&] {

      RowWiseAdaGradVecOptimizer<g_t, w_t> opt {lr, eps};
      if (dim % 4 == 0) {
        const int max_grid_size = device_prop.num_sms * (device_prop.max_thread_per_sm / OPTIMIZER_BLOCKSIZE_VEC);
        const int warp_per_block = OPTIMIZER_BLOCKSIZE_VEC / WARPSIZE;

        int grid_size = 0;
        if (ev_nums / warp_per_block < max_grid_size) {
          grid_size = (ev_nums-1) / warp_per_block + 1;
        }
        else if (ev_nums / warp_per_block > max_grid_size * MULTIPLIER) {
          grid_size = max_grid_size * MULTIPLIER;
        } else {
          grid_size = max_grid_size;
        }

        auto kernel = update4_kernel<g_t, w_t, decltype(opt)>;
        kernel<<<grid_size, OPTIMIZER_BLOCKSIZE_VEC, 0, stream>>>(
          ev_nums, dim, reinterpret_cast<const g_t *>(grads.data_ptr()),
          reinterpret_cast<w_t **>(vector_ptrs.data_ptr()), founds.data_ptr<bool>(), opt);
        DEMB_CUDA_KERNEL_LAUNCH_CHECK();

      } else {

        int block_size = dim > OPTIMIZER_BLOCKSIZE ? OPTIMIZER_BLOCKSIZE : dim;
        int grid_size = ev_nums;
        int shared_memory_bytes = block_size * sizeof(float);

        auto kernel = update_kernel<g_t, w_t, decltype(opt)>;
        kernel<<<grid_size, block_size, shared_memory_bytes, stream>>>(
          ev_nums, dim, reinterpret_cast<const g_t *>(grads.data_ptr()),
          reinterpret_cast<w_t **>(vector_ptrs.data_ptr()), founds.data_ptr<bool>(), opt);
        DEMB_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  });
  DEMB_CUDA_KERNEL_LAUNCH_CHECK();
}

} // namespace dyn_emb

// PYTHON WRAP
void bind_optimizer_kernel_op(py::module &m) {
  m.def("dynamic_emb_sgd_with_table", &dyn_emb::dynamic_emb_sgd_with_table,
        "SGD optimizer for Dynamic Emb", py::arg("table"),
        py::arg("n"), py::arg("indices"), py::arg("grads"),
        py::arg("lr"), py::arg("weight_type"), py::arg("score") = py::none(), py::arg("emb") = c10::nullopt,
        py::arg("host_table") = py::none());

  m.def("dynamic_emb_adam_with_table", &dyn_emb::dynamic_emb_adam_with_table,
        "Adam optimizer for Dynamic Emb", py::arg("ht"),
        py::arg("n"), py::arg("indices"), py::arg("grads"),
        py::arg("lr"), py::arg("beta1"),
        py::arg("beta2"), py::arg("eps"), py::arg("weight_decay"), py::arg("iter_num"),
        py::arg("weight_type"), py::arg("score") = py::none(), py::arg("emb") = c10::nullopt,
        py::arg("host_table") = py::none());

  m.def("dynamic_emb_adagrad_with_table", &dyn_emb::dynamic_emb_adagrad_with_table,
        "Adagrad optimizer for Dynamic Emb", py::arg("ht"),
        py::arg("n"), py::arg("indices"), py::arg("grads"),py::arg("lr"),
        py::arg("eps"),
        py::arg("weight_type"), py::arg("score") = py::none(), py::arg("emb") = c10::nullopt,
        py::arg("host_table") = py::none());

  m.def("dynamic_emb_rowwise_adagrad_with_table", &dyn_emb::dynamic_emb_rowwise_adagrad_with_table,
        "Row Wise Adagrad optimizer for Dynamic Emb", py::arg("ht"),
        py::arg("n"), py::arg("indices"), py::arg("grads"),py::arg("lr"),
        py::arg("eps"),
        py::arg("weight_type"), py::arg("score") = py::none(), py::arg("emb") = c10::nullopt,
        py::arg("host_table") = py::none());
}
