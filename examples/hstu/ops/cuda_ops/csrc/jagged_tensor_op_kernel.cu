#include "hip/hip_runtime.h"
#include <ATen/Functions.h>
#include <torch/torch.h>
#include <torch/extension.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <c10/cuda/CUDAException.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include "../include/utils.h"

constexpr int kMaxNumTensors = 128;
template <typename T>
struct InputJaggedTensor {
	T* value_list[kMaxNumTensors];
	int32_t* offsets_list[kMaxNumTensors];
};

// float4 vectorized copy function
template<typename T>
__device__ __forceinline__ void copy_float4(T* dst, const T* src) {
    if (sizeof(T) == sizeof(float) && 
        reinterpret_cast<uintptr_t>(dst) % 16 == 0 && 
        reinterpret_cast<uintptr_t>(src) % 16 == 0) {
        // float4 copy
        *reinterpret_cast<float4*>(dst) = *reinterpret_cast<const float4*>(src);
    } else if (sizeof(T) == sizeof(at::Half) && 
               reinterpret_cast<uintptr_t>(dst) % 16 == 0 && 
               reinterpret_cast<uintptr_t>(src) % 16 == 0) {
        // half8 copy (8 half = 1 float4)
        *reinterpret_cast<float4*>(dst) = *reinterpret_cast<const float4*>(src);
    } else {
        // fallback to scalar copy
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            dst[i] = src[i];
        }
    }
}

template <typename T>
__launch_bounds__(1024, 2) __global__ void concat_2D_jagged_tensors_forward_kernel_opt_v4(
// __global__ void concat_2D_jagged_tensors_forward_kernel_opt_v4(
    const InputJaggedTensor<T> input_jagged_tensor,
    const int32_t num_tensors,
    const int32_t batch_size,
    const int32_t hidden_dim,
    const int32_t max_seqlen,
    const int32_t total_blocks,
    const int32_t seqlen_per_block,
    int* workload_offset,
    T* merged_values,
    int* merged_offsets) {
    // each block processes workload_offset[i+1]-workload_offset[i] sequences
    for(int block_id = blockIdx.x; block_id < total_blocks; block_id += gridDim.x){
        if(workload_offset[block_id] == workload_offset[block_id + 1]) continue;
        
        int num_bucket_per_batch = (max_seqlen + seqlen_per_block - 1) / seqlen_per_block;
        int tensor_id = (block_id / num_bucket_per_batch) % num_tensors;
        int batch_id =  block_id / num_bucket_per_batch / num_tensors; 
        int idx = block_id % num_bucket_per_batch; // which bucket
        int warp_id = threadIdx.x / 32;
        int lane_id = threadIdx.x % 32;
        const int32_t* offsets = input_jagged_tensor.offsets_list[tensor_id];
        const T* values = input_jagged_tensor.value_list[tensor_id];
        int seq_start = offsets[batch_id];
        
        // for merged_values, each block processes workload_offset[block_id+1]-workload_offset[block_id] sequences starting from workload_offset[block_id] 
        // for source_values, copy len sequences from input_jagged_tensor.value_list[tensor_id] starting from seq_start

        for(int seq_offset = warp_id; seq_offset < workload_offset[block_id+1]-workload_offset[block_id]; seq_offset += 32){
            int src_row = seq_start + seq_offset + idx*seqlen_per_block; // add bucket offset
            int dst_row = workload_offset[block_id] + seq_offset;
            // warp-level parallelization for hidden_dim
            // each thread in warp handles multiple hidden dimensions
            int elements_per_thread = (hidden_dim + 32 - 1) / 32;
            int thread_start = lane_id * elements_per_thread;
            int thread_end = min(thread_start + elements_per_thread, hidden_dim);

            // vectorized copy 
            if (hidden_dim % 4 == 0 && thread_start % 4 == 0) {
                int vectorized_start = (thread_start + 3) / 4 * 4;
                int vectorized_end = thread_end / 4 * 4;
                
                for (int h = thread_start; h < vectorized_start && h < thread_end; ++h) {
                    merged_values[dst_row * hidden_dim + h] = values[src_row * hidden_dim + h];
                }
                
                for (int h = vectorized_start; h < vectorized_end; h += 4) {
                    copy_float4(&merged_values[dst_row * hidden_dim + h],
                                &values[src_row * hidden_dim + h]);
                }
                
                // handle remaining elements
                for (int h = vectorized_end; h < thread_end; ++h) {
                    merged_values[dst_row * hidden_dim + h] = values[src_row * hidden_dim + h];
                }
            } else {
                // scalar copy
                for (int h = thread_start; h < thread_end; ++h) {
                    merged_values[dst_row * hidden_dim + h] = values[src_row * hidden_dim + h];
                }
            }
        }
    }
}


template <typename T, int HIDDEN_DIM>
__launch_bounds__(1024, 2) __global__ void concat_2D_jagged_tensors_forward_kernel_alignment(
    const InputJaggedTensor<T> input_jagged_tensor,
    const int32_t num_tensors,
    const int32_t batch_size,
    const int32_t max_seqlen,
    const int32_t total_blocks,
    const int32_t seqlen_per_block,
    int* workload_offset,
    T* merged_values,
    int* merged_offsets) {
    // each block processes workload_offset[i+1]-workload_offset[i] sequences
    constexpr int elements_per_thread = (HIDDEN_DIM + 31) / 32;
    for(int block_id = blockIdx.x; block_id < total_blocks; block_id += gridDim.x){
        if(workload_offset[block_id] == workload_offset[block_id + 1]) continue;
        
        int num_bucket_per_batch = (max_seqlen + seqlen_per_block - 1) / seqlen_per_block;
        int tensor_id = (block_id / num_bucket_per_batch) % num_tensors;
        int batch_id =  block_id / num_bucket_per_batch / num_tensors; 
        int idx = block_id % num_bucket_per_batch; // which bucket
        int warp_id = threadIdx.x / 32;
        int lane_id = threadIdx.x % 32;
        const int32_t* offsets = input_jagged_tensor.offsets_list[tensor_id];
        const T* values = input_jagged_tensor.value_list[tensor_id];
        int seq_start = offsets[batch_id];
        
        // for merged_values, each block processes workload_offset[block_id+1]-workload_offset[block_id] sequences starting from workload_offset[block_id] 
        // for source_values, copy len sequences from input_jagged_tensor.value_list[tensor_id] starting from seq_start

        for(int seq_offset = warp_id; seq_offset < workload_offset[block_id+1]-workload_offset[block_id]; seq_offset += 32){
            int src_row = seq_start + seq_offset + idx*seqlen_per_block; // add bucket offset
            int dst_row = workload_offset[block_id] + seq_offset;
            int h_start = lane_id * elements_per_thread;
            #pragma unroll
            for(int i = 0; i < elements_per_thread; i += 4) {
                copy_float4(&merged_values[dst_row * HIDDEN_DIM + h_start + i], &values[src_row * HIDDEN_DIM + h_start + i]);
            }
        }
    }
}
template <typename T>
__launch_bounds__(1024, 2) __global__ void concat_2D_jagged_tensors_forward_kernel_warp(
    const InputJaggedTensor<T> input_jagged_tensor,
    const int32_t num_tensors,
    const int32_t batch_size,
    const int32_t hidden_dim,
    const int32_t max_seqlen,
    const int32_t total_blocks,
    const int32_t seqlen_per_block,
    int* workload_offset,
    T* merged_values,
    int* merged_offsets) {
    // each block processes workload_offset[i+1]-workload_offset[i] sequences
    for(int block_id = blockIdx.x; block_id < total_blocks; block_id += gridDim.x){
        if(workload_offset[block_id] == workload_offset[block_id + 1]) continue;
        
        int num_bucket_per_batch = (max_seqlen + seqlen_per_block - 1) / seqlen_per_block;
        int tensor_id = (block_id / num_bucket_per_batch) % num_tensors;
        int batch_id =  block_id / num_bucket_per_batch / num_tensors; 
        int idx = block_id % num_bucket_per_batch; // which bucket
        int warp_id = threadIdx.x / 32;
        int lane_id = threadIdx.x % 32;
        const int32_t* offsets = input_jagged_tensor.offsets_list[tensor_id];
        const T* values = input_jagged_tensor.value_list[tensor_id];
        int seq_start = offsets[batch_id];
        
        // for merged_values, each block processes workload_offset[block_id+1]-workload_offset[block_id] sequences starting from workload_offset[block_id] 
        // for source_values, copy len sequences from input_jagged_tensor.value_list[tensor_id] starting from seq_start

        for(int seq_offset = warp_id; seq_offset < workload_offset[block_id+1]-workload_offset[block_id]; seq_offset += 32){
            int src_row = seq_start + seq_offset + idx*seqlen_per_block; // add bucket offset
            int dst_row = workload_offset[block_id] + seq_offset;
            int vec4_count = hidden_dim / 4;
            for (int i = lane_id; i < vec4_count; i += 32) {
                int h = i * 4;
                copy_float4(&merged_values[dst_row * hidden_dim + h], &values[src_row * hidden_dim + h]);
            }
        }
    }
}
void concat_2D_jagged_tensors_cuda_forward (
    const std::vector<torch::Tensor>& values_list,
    const std::vector<torch::Tensor>& offsets_list,
    int seqlen_per_block,
    int max_seqlen,
    int max_block_size,
    torch::Tensor workload_offset,
    torch::Tensor merged_values,
    torch::Tensor merged_offsets){

    int num_tensors = values_list.size();
    if (num_tensors == 0) {
        return; 
    }

    int hidden_dim = values_list[0].size(-1);

    int batch_size = offsets_list[0].size(0) - 1;


    assert(merged_values.is_contiguous());

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::kHalf, at::kBFloat16,
        values_list[0].scalar_type(), 
        "concat_2D_jagged_tensors_forward_kernel",
        [&] {
            InputJaggedTensor<scalar_t> input_jagged_tensor_typed;
            for (int i = 0; i < num_tensors; ++i) {
                TORCH_CHECK(i < kMaxNumTensors, "Number of tensors exceeds kMaxNumTensors");
                input_jagged_tensor_typed.value_list[i] = values_list[i].data_ptr<scalar_t>();
                input_jagged_tensor_typed.offsets_list[i] = offsets_list[i].data_ptr<int32_t>();
            }
            int blocks_per_batch = (max_seqlen + seqlen_per_block - 1) / seqlen_per_block;
            int total_blocks = batch_size * blocks_per_batch * num_tensors;

            // warp configuration: ensure not exceeding 1024 threads, each warp processes 1 sequence
            int target_warps = min(32, max(1, seqlen_per_block)); 
            int threads = min(1024, target_warps * 32);
            //todo:python side max_grid_size not work now, use hipDeviceProp_t to get max_grid_size
            // hipDeviceProp_t prop;
            // hipGetDeviceProperties(&prop, 0);
            // int max_grid_size = prop.maxGridSize[0];

            dim3 opt_blocks(min(max_block_size, total_blocks));
            // dim3 opt_blocks(total_blocks);
            dim3 opt_threads(threads);
            
            int elements_per_thread = (hidden_dim + 32 - 1) / 32;
            if(hidden_dim % 4 == 0 && elements_per_thread % 4 == 0){
                switch(hidden_dim){
                    case 128:
                        concat_2D_jagged_tensors_forward_kernel_alignment<scalar_t, 128><<<opt_blocks, opt_threads, 0, stream>>>(
                            input_jagged_tensor_typed,
                            num_tensors,
                            batch_size,
                            max_seqlen,
                            total_blocks,
                            seqlen_per_block,
                            workload_offset.data_ptr<int>(),
                            merged_values.data_ptr<scalar_t>(),
                            merged_offsets.data_ptr<int>()
                        );
                        break;
                    case 256:
                        concat_2D_jagged_tensors_forward_kernel_alignment<scalar_t, 256><<<opt_blocks, opt_threads, 0, stream>>>(
                            input_jagged_tensor_typed,
                            num_tensors,
                            batch_size,
                            max_seqlen,
                            total_blocks,
                            seqlen_per_block,
                            workload_offset.data_ptr<int>(),
                            merged_values.data_ptr<scalar_t>(),
                            merged_offsets.data_ptr<int>()
                        );
                        break;
                    case 512:
                        concat_2D_jagged_tensors_forward_kernel_alignment<scalar_t, 512><<<opt_blocks, opt_threads, 0, stream>>>(
                            input_jagged_tensor_typed,
                            num_tensors,
                            batch_size,
                            max_seqlen,
                            total_blocks,
                            seqlen_per_block,
                            workload_offset.data_ptr<int>(),
                            merged_values.data_ptr<scalar_t>(),
                            merged_offsets.data_ptr<int>()
                        );
                        break;
                    default:
                        concat_2D_jagged_tensors_forward_kernel_warp<scalar_t><<<opt_blocks, opt_threads, 0, stream>>>(
                            input_jagged_tensor_typed,
                            num_tensors,
                            batch_size,
                            hidden_dim,
                            max_seqlen,
                            total_blocks,
                            seqlen_per_block,
                            workload_offset.data_ptr<int>(),
                            merged_values.data_ptr<scalar_t>(),
                            merged_offsets.data_ptr<int>()
                        );
                        break;
                }

            }
            else if(hidden_dim % 4 == 0){
                concat_2D_jagged_tensors_forward_kernel_warp<scalar_t><<<opt_blocks, opt_threads, 0, stream>>>(
                    input_jagged_tensor_typed,
                    num_tensors,
                    batch_size,
                    hidden_dim,
                    max_seqlen,
                    total_blocks,
                    seqlen_per_block,
                    workload_offset.data_ptr<int>(),
                    merged_values.data_ptr<scalar_t>(),
                    merged_offsets.data_ptr<int>()
                );
            }
            else {
                concat_2D_jagged_tensors_forward_kernel_opt_v4<scalar_t><<<opt_blocks, opt_threads, 0, stream>>>(
                    input_jagged_tensor_typed,
                    num_tensors,
                    batch_size,
                    hidden_dim,
                    max_seqlen,
                    total_blocks,
                    seqlen_per_block,
                    workload_offset.data_ptr<int>(),
                    merged_values.data_ptr<scalar_t>(),
                    merged_offsets.data_ptr<int>()
                );
            }

            
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        }
    );

    return; 
}

template <typename T>
__global__ void concat_2D_jagged_tensors_backward_kernel(
    const InputJaggedTensor<T> grad_jagged_tensor,
    const int32_t num_tensors,
    const int32_t num_rows,
    const int32_t hidden_dim,
    const T* grad_output,
    int* merged_offsets) {
    
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= num_rows) return;
    int out_idx = merged_offsets[row];

    for (int t = 0; t < num_tensors; ++t) {
        T* grad_values = grad_jagged_tensor.value_list[t];
        const int32_t* offsets = grad_jagged_tensor.offsets_list[t];
        int start = offsets[row];
        int end = offsets[row + 1];
        for (int i = start; i < end; ++i) {
            for (int h = 0; h < hidden_dim; ++h) {
                grad_values[i * hidden_dim + h] = grad_output[out_idx * hidden_dim + h];
            }
            out_idx++;
        }
    }
}

template <typename T>
__global__ void concat_2D_jagged_tensors_backward_kernel_opt(
    const InputJaggedTensor<T> grad_jagged_tensor,
    const int32_t num_tensors,
    const int32_t num_rows,
    const int32_t hidden_dim,
    const T* grad_output,
    int* merged_offsets) {
    
    __shared__ int32_t shared_lens[kMaxNumTensors];
    
    int batch_id = blockIdx.x;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    
    if (batch_id >= num_rows) return;
    
    if (threadIdx.x < num_tensors) {
        const int32_t* offsets = grad_jagged_tensor.offsets_list[threadIdx.x];
        shared_lens[threadIdx.x] = offsets[batch_id + 1] - offsets[batch_id];
    }
    __syncthreads();
    
    if (warp_id < num_tensors) {
        T* grad_values = grad_jagged_tensor.value_list[warp_id];
        const int32_t* offsets = grad_jagged_tensor.offsets_list[warp_id];
        int start = offsets[batch_id];
        int end = offsets[batch_id + 1];
        int num_rows_in_tensor = end - start;
        
        int out_start = merged_offsets[batch_id];
        for (int t = 0; t < warp_id; ++t) {
            out_start += shared_lens[t];
        }
        
        for (int row_offset = lane_id; row_offset < num_rows_in_tensor; row_offset += 32) {
            int i = start + row_offset;
            int out_row = out_start + row_offset;
            //todo: split here to two kernels
            if (hidden_dim % 4 == 0 && hidden_dim <= 256) {
                for (int h = 0; h < hidden_dim; h += 4) {
                    copy_float4(&grad_values[i * hidden_dim + h],
                               &grad_output[out_row * hidden_dim + h]);
                }
            } else {
                for (int h = 0; h < hidden_dim; ++h) {
                    grad_values[i * hidden_dim + h] = grad_output[out_row * hidden_dim + h];
                }
            }
        }
    }
}
template <typename T>
__launch_bounds__(1024, 2) __global__ void concat_2D_jagged_tensors_backward_kernel_opt_v4(
    const InputJaggedTensor<T> grad_jagged_tensor,
    const int32_t num_tensors,
    const int32_t batch_size,
    const int32_t hidden_dim,
    const int32_t max_seqlen,
    const int32_t total_blocks,
    const int32_t seqlen_per_block,
    int* workload_offset,
    T* grad_output,
    int* merged_offsets) {
    // each block processes workload_offset[i+1]-workload_offset[i] sequences
    for(int block_id = blockIdx.x; block_id < total_blocks; block_id += gridDim.x){
        if(workload_offset[block_id] == workload_offset[block_id + 1]) continue;
        
        int num_bucket_per_batch = (max_seqlen + seqlen_per_block - 1) / seqlen_per_block;
        int tensor_id = (block_id / num_bucket_per_batch) % num_tensors;
        int batch_id =  block_id / num_bucket_per_batch / num_tensors; 
        int idx = block_id % num_bucket_per_batch; // which bucket
        int warp_id = threadIdx.x / 32;
        int lane_id = threadIdx.x % 32;
        const int32_t* offsets = grad_jagged_tensor.offsets_list[tensor_id];
        T* values = grad_jagged_tensor.value_list[tensor_id];
        int seq_start = offsets[batch_id];
        int seq_end = offsets[batch_id + 1];
        int len = seq_end - seq_start;

        for(int seq_offset = warp_id; seq_offset < workload_offset[block_id+1]-workload_offset[block_id]; seq_offset += 32){
            // add bucket offset
            int src_row = seq_start + seq_offset + idx*seqlen_per_block; 
            int dst_row = workload_offset[block_id] + seq_offset;
            // warp-level parallelization for hidden_dim
            // each thread in warp handles multiple hidden dimensions
            int elements_per_thread = (hidden_dim + 32 - 1) / 32;
            int thread_start = lane_id * elements_per_thread;
            int thread_end = min(thread_start + elements_per_thread, hidden_dim);

            // vectorized copy 
            if (hidden_dim % 4 == 0 && thread_start % 4 == 0) {
                int vectorized_start = (thread_start + 3) / 4 * 4;
                int vectorized_end = thread_end / 4 * 4;
                
                for (int h = thread_start; h < vectorized_start && h < thread_end; ++h) {
                    values[src_row * hidden_dim + h] = grad_output[dst_row * hidden_dim + h];
                }
                
                for (int h = vectorized_start; h < vectorized_end; h += 4) {
                    copy_float4(&values[src_row * hidden_dim + h],
                                &grad_output[dst_row * hidden_dim + h]);
                }
                
                // handle remaining elements
                for (int h = vectorized_end; h < thread_end; ++h) {
                    values[src_row * hidden_dim + h] = grad_output[dst_row * hidden_dim + h];
                }
            } else {
                // scalar copy
                for (int h = thread_start; h < thread_end; ++h) {
                    values[src_row * hidden_dim + h] = grad_output[dst_row * hidden_dim + h];
                }
            }
        }
    }
}
void concat_2D_jagged_tensors_cuda_backward(
    torch::Tensor grad_output,
    torch::Tensor grad_lengths,  
    int seqlen_per_block,
    int max_seqlen,
    int max_block_size,
    torch::Tensor workload_offset,
    const std::vector<torch::Tensor>& grad_inputs,
    const std::vector<torch::Tensor>& offsets_list,
    torch::Tensor merged_offsets) {

    int num_tensors = offsets_list.size();
    int batch_size = grad_lengths.size(0);
    int hidden_dim = grad_output.size(-1);

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::kHalf, at::kBFloat16,
        grad_output.scalar_type(), 
        "concat_2D_jagged_tensors_backward_kernel",
        [&] {
            InputJaggedTensor<scalar_t> grad_jagged_tensor;
            for (int i = 0; i < num_tensors; ++i) {
                TORCH_CHECK(i < kMaxNumTensors, "Number of tensors exceeds kMaxNumTensors");
                grad_jagged_tensor.value_list[i] = grad_inputs[i].data_ptr<scalar_t>();
                grad_jagged_tensor.offsets_list[i] = offsets_list[i].data_ptr<int32_t>();
            }
            int blocks_per_batch = (max_seqlen + seqlen_per_block - 1) / seqlen_per_block;
            int total_blocks = batch_size * blocks_per_batch * num_tensors;
            
            // warp configuration: ensure not exceeding 1024 threads, each warp processes 1 sequence
            int target_warps = min(32, max(1, seqlen_per_block)); 
            int threads = min(1024, target_warps * 32);
            //todo:python side max_grid_size not work now, use hipDeviceProp_t to get max_grid_size
            // hipDeviceProp_t prop;
            // hipGetDeviceProperties(&prop, 0);
            // int max_grid_size = prop.maxGridSize[0];
            dim3 opt_blocks(min(max_block_size, total_blocks));
            dim3 opt_threads(threads);
            concat_2D_jagged_tensors_backward_kernel_opt_v4<scalar_t><<<opt_blocks, opt_threads, 0, stream>>>(
                grad_jagged_tensor,
                num_tensors,
                batch_size,
                hidden_dim,
                max_seqlen,
                total_blocks,
                seqlen_per_block,
                workload_offset.data_ptr<int>(),
                grad_output.data_ptr<scalar_t>(),
                merged_offsets.data_ptr<int>()
            );
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        }
    );
    return;
}

__global__ void compute_block_workloads_cuda_kernel(
    const InputJaggedTensor<int32_t> input_jagged_tensor,
    const int32_t num_tensors,
    const int32_t batch_size,
    const int32_t seqlen_per_block,
    const int32_t max_seqlen,
    const int32_t total_blocks,
    int* block_workloads) {
    
    int work_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (work_id >= total_blocks) return;
    
    int num_bucket_per_batch = (max_seqlen + seqlen_per_block - 1) / seqlen_per_block;

    int tensor_id = (work_id / num_bucket_per_batch) % num_tensors;
    int batch_id = work_id / num_bucket_per_batch / num_tensors; 
    int idx = work_id % num_bucket_per_batch; // which bucket
    const int32_t* offsets = input_jagged_tensor.offsets_list[tensor_id];
    int seq_start = offsets[batch_id];
    int seq_end = offsets[batch_id + 1];
    int len = seq_end - seq_start;

    int remaining_len = len - idx * seqlen_per_block;
    block_workloads[work_id] = max(0, min(remaining_len, seqlen_per_block));

    return;
}

/*
target: determine how many rows each block should process from merged_offsets
e.g. [4, 1, 4, 4, 4, 0, 4, 1, ...]

Working principle:
1. Each block is responsible for one bucket (sequence segment of seqlen_per_block size) of a batch
2. Calculate the total number of sequences for all tensors in that batch
3. Calculate the actual workload for that block based on bucket position
4. Output to block_workloads array for subsequent prefix sum calculation
*/
void compute_block_workloads_cuda(
    const std::vector<torch::Tensor>& offsets_list,
    int seqlen_per_block,
    int max_seqlen,
    torch::Tensor block_workloads) {

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    int num_tensors = offsets_list.size();
    int batch_size = offsets_list[0].size(0) - 1;
    int blocks_per_batch = (max_seqlen + seqlen_per_block - 1) / seqlen_per_block;
    int total_blocks = batch_size * blocks_per_batch * num_tensors;

    InputJaggedTensor<int32_t> offsets_jagged_tensor;
    for (int i = 0; i < num_tensors; ++i) {
        TORCH_CHECK(i < kMaxNumTensors, "Number of tensors exceeds kMaxNumTensors");
        offsets_jagged_tensor.offsets_list[i] = offsets_list[i].data_ptr<int32_t>();
    }
    int threads_per_block = min(1024, total_blocks);
    int num_blocks = (total_blocks + threads_per_block - 1) / threads_per_block;
    dim3 blocks(num_blocks);
    dim3 threads(threads_per_block);
    
    compute_block_workloads_cuda_kernel<<<blocks, threads, 0, stream>>>(
        offsets_jagged_tensor,
        num_tensors,
        batch_size,
        seqlen_per_block,
        max_seqlen,
        total_blocks,
        block_workloads.data_ptr<int>()
    );
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    return;
}