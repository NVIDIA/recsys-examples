#include "hip/hip_runtime.h"
#include <ATen/Functions.h>
#include <torch/torch.h>
#include <torch/extension.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <c10/cuda/CUDAException.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include "../include/utils.h"
constexpr int kMaxNumTensors = 32;
template <typename T>
struct InputJaggedTensor {
	T* value_list[kMaxNumTensors];
	int32_t* offsets_list[kMaxNumTensors];
};

// float4 vectorized copy function
template<typename T>
__device__ __forceinline__ void copy_float4(T* dst, const T* src) {
    if (sizeof(T) == sizeof(float) && 
        reinterpret_cast<uintptr_t>(dst) % 16 == 0 && 
        reinterpret_cast<uintptr_t>(src) % 16 == 0) {
        // float4 copy
        *reinterpret_cast<float4*>(dst) = *reinterpret_cast<const float4*>(src);
    } else if (sizeof(T) == sizeof(at::Half) && 
               reinterpret_cast<uintptr_t>(dst) % 16 == 0 && 
               reinterpret_cast<uintptr_t>(src) % 16 == 0) {
        // half8 copy (8 half = 1 float4)
        *reinterpret_cast<float4*>(dst) = *reinterpret_cast<const float4*>(src);
    } else {
        // fallback to scalar copy
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            dst[i] = src[i];
        }
    }
}

template <typename T>
__global__ void concat_2D_jagged_tensors_forward_kernel(
	const InputJaggedTensor<T> input_jagged_tensor,
	const int32_t num_tensors,
	const int32_t num_rows,
	const int32_t hidden_dim,
	T* merged_values,
    int* merged_offsets) {
	
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows) return;
	int out_idx = merged_offsets[row];

	for (int t = 0; t < num_tensors; ++t) {
		const T* values = input_jagged_tensor.value_list[t];
		const int32_t* offsets = input_jagged_tensor.offsets_list[t];
		int start = offsets[row];
		int end = offsets[row + 1];

		for (int i = start; i < end; ++i) {
			for (int h = 0; h < hidden_dim; ++h) {
				merged_values[out_idx * hidden_dim + h] = values[i * hidden_dim + h];
			}
			out_idx++;
		} 
	}
}

template <typename T>
__global__ void concat_2D_jagged_tensors_forward_kernel_opt(
	const InputJaggedTensor<T> input_jagged_tensor,
	const int32_t num_tensors,
	const int32_t num_rows,
	const int32_t hidden_dim,
	T* merged_values,
    int* merged_offsets) {
	
    // shared memory to cache offset lengths for current batch
    __shared__ int32_t shared_lens[kMaxNumTensors];
    
    int batch_id = blockIdx.x;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    
    if (batch_id >= num_rows) return;
    
    // load offset lengths to shared memory
    if (threadIdx.x < num_tensors) {
        const int32_t* offsets = input_jagged_tensor.offsets_list[threadIdx.x];
        shared_lens[threadIdx.x] = offsets[batch_id + 1] - offsets[batch_id];
    }
    __syncthreads();
    
    // each warp processes one tensor
    if (warp_id < num_tensors) {
        const T* values = input_jagged_tensor.value_list[warp_id];
        const int32_t* offsets = input_jagged_tensor.offsets_list[warp_id];
        int start = offsets[batch_id];
        int end = offsets[batch_id + 1];
        int num_rows_in_tensor = end - start;
        
        // calculate output start position for current tensor
        int out_start = merged_offsets[batch_id];
        for (int t = 0; t < warp_id; ++t) {
            out_start += shared_lens[t];
        }

        //each thread handles different rows
        for (int row_offset = lane_id; row_offset < num_rows_in_tensor; row_offset += 32) {
            int i = start + row_offset;
            int out_row = out_start + row_offset;
            
            // each thread copies entire hidden_dim 
            if (hidden_dim % 4 == 0 && hidden_dim <= 256) {
                for (int h = 0; h < hidden_dim; h += 4) {
                    copy_float4(&merged_values[out_row * hidden_dim + h],
                               &values[i * hidden_dim + h]);
                }
            } else {
                for (int h = 0; h < hidden_dim; ++h) {
                    merged_values[out_row * hidden_dim + h] = values[i * hidden_dim + h];
                }
            }
        }
    }
}

__global__ void concat_1D_jagged_tensor_kernel(
	const float** values_list,
	const int** offsets_list,
	int num_tensor,
	int num_rows,//total_length
	float* merged_values,
	int* merged_offsets){
	
    int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows) return;

	int out_idx = merged_offsets[row]; // data start from this row
	for(int i = 0; i < num_tensor; i++){
		const float* values = values_list[i];
		const int* offsets = offsets_list[i];
		int st = offsets[row];
		int end = offsets[row+1];
		for(int j = st; j < end; j++){
			merged_values[out_idx++] = values[j];
		}
	}
}

void concat_2D_jagged_tensors_cuda_forward (
    const std::vector<torch::Tensor>& values_list,
    const std::vector<torch::Tensor>& offsets_list,
    torch::Tensor merged_values,
    torch::Tensor merged_offsets){

    int num_tensors = values_list.size();
    if (num_tensors == 0) {
        return; 
    }
    int num_rows = offsets_list[0].size(0) - 1;
    int hidden_dim = values_list[0].size(-1);

    int threads = 128;
    int blocks = (num_rows + threads - 1) / threads;

    assert(merged_values.is_contiguous());

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    // printf("values_list[0].scalar_type() = %d\n", values_list[0].scalar_type());
    // DISPATCH_KERNEL_BY_TYPE(
    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::kHalf, at::kBFloat16,
        values_list[0].scalar_type(), 
        "concat_2D_jagged_tensors_forward_kernel",
        [&] {
            InputJaggedTensor<scalar_t> input_jagged_tensor_typed;
            for (int i = 0; i < num_tensors; ++i) {
                TORCH_CHECK(i < kMaxNumTensors, "Number of tensors exceeds kMaxNumTensors");
                input_jagged_tensor_typed.value_list[i] = values_list[i].data_ptr<scalar_t>();
                input_jagged_tensor_typed.offsets_list[i] = offsets_list[i].data_ptr<int32_t>();
            }

            if (hidden_dim <= 256 && num_tensors <= 16) {
                dim3 opt_blocks(num_rows);
                dim3 opt_threads(min(num_tensors * 32, 1024));  // each tensor gets one warp
                
                concat_2D_jagged_tensors_forward_kernel_opt<scalar_t><<<opt_blocks, opt_threads, 0, stream>>>(
                    input_jagged_tensor_typed,
                    num_tensors,
                    num_rows,
                    hidden_dim,
                    merged_values.data_ptr<scalar_t>(),
                    merged_offsets.data_ptr<int>()
                );
            } else {
                // use old kernel for large problems(for now)
                // Todo: optimize this kernel
                concat_2D_jagged_tensors_forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
                    input_jagged_tensor_typed,
                    num_tensors,
                    num_rows,
                    hidden_dim,
                    merged_values.data_ptr<scalar_t>(),
                    merged_offsets.data_ptr<int>()
                );
            }
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        }
    );

    return; 
}

template <typename T>
__global__ void concat_2D_jagged_tensors_backward_kernel(
    const InputJaggedTensor<T> grad_jagged_tensor,
    const int32_t num_tensors,
    const int32_t num_rows,
    const int32_t hidden_dim,
    const T* grad_output,
    int* merged_offsets) {
    
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= num_rows) return;
    int out_idx = merged_offsets[row];

    for (int t = 0; t < num_tensors; ++t) {
        T* grad_values = grad_jagged_tensor.value_list[t];
        const int32_t* offsets = grad_jagged_tensor.offsets_list[t];
        int start = offsets[row];
        int end = offsets[row + 1];
        for (int i = start; i < end; ++i) {
            for (int h = 0; h < hidden_dim; ++h) {
                grad_values[i * hidden_dim + h] = grad_output[out_idx * hidden_dim + h];
            }
            out_idx++;
        }
    }
}

template <typename T>
__global__ void concat_2D_jagged_tensors_backward_kernel_opt(
    const InputJaggedTensor<T> grad_jagged_tensor,
    const int32_t num_tensors,
    const int32_t num_rows,
    const int32_t hidden_dim,
    const T* grad_output,
    int* merged_offsets) {
    
    __shared__ int32_t shared_lens[kMaxNumTensors];
    
    int batch_id = blockIdx.x;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    
    if (batch_id >= num_rows) return;
    
    if (threadIdx.x < num_tensors) {
        const int32_t* offsets = grad_jagged_tensor.offsets_list[threadIdx.x];
        shared_lens[threadIdx.x] = offsets[batch_id + 1] - offsets[batch_id];
    }
    __syncthreads();
    
    if (warp_id < num_tensors) {
        T* grad_values = grad_jagged_tensor.value_list[warp_id];
        const int32_t* offsets = grad_jagged_tensor.offsets_list[warp_id];
        int start = offsets[batch_id];
        int end = offsets[batch_id + 1];
        int num_rows_in_tensor = end - start;
        
        int out_start = merged_offsets[batch_id];
        for (int t = 0; t < warp_id; ++t) {
            out_start += shared_lens[t];
        }
        
        for (int row_offset = lane_id; row_offset < num_rows_in_tensor; row_offset += 32) {
            int i = start + row_offset;
            int out_row = out_start + row_offset;
            
            if (hidden_dim % 4 == 0 && hidden_dim <= 256) {
                for (int h = 0; h < hidden_dim; h += 4) {
                    copy_float4(&grad_values[i * hidden_dim + h],
                               &grad_output[out_row * hidden_dim + h]);
                }
            } else {
                for (int h = 0; h < hidden_dim; ++h) {
                    grad_values[i * hidden_dim + h] = grad_output[out_row * hidden_dim + h];
                }
            }
        }
    }
}

std::vector<torch::Tensor> concat_2D_jagged_tensors_cuda_backward(
    torch::Tensor grad_output,
    torch::Tensor grad_lengths,
    const std::vector<torch::Tensor>& offsets_list,
    torch::Tensor merged_offsets) {

    int num_tensors = offsets_list.size();
    int num_rows = grad_lengths.size(0);
    int hidden_dim = grad_output.size(-1);

    std::vector<torch::Tensor> grad_inputs(num_tensors);
    for (int i = 0; i < num_tensors; ++i) {
        int tensor_size = offsets_list[i].index({offsets_list[i].size(0) - 1}).item<int>();
        grad_inputs[i] = torch::empty(
            {tensor_size, hidden_dim},
            grad_output.options()
        );
    }

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    int threads = 128;
    int blocks = (num_rows + threads - 1) / threads;


    // DISPATCH_KERNEL_BY_TYPE(
    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::kHalf, at::kBFloat16,
        grad_output.scalar_type(), 
        "concat_2D_jagged_tensors_backward_kernel",
        [&] {
            InputJaggedTensor<scalar_t> grad_jagged_tensor;
            for (int i = 0; i < num_tensors; ++i) {
                TORCH_CHECK(i < kMaxNumTensors, "Number of tensors exceeds kMaxNumTensors");
                grad_jagged_tensor.value_list[i] = grad_inputs[i].data_ptr<scalar_t>();
                grad_jagged_tensor.offsets_list[i] = offsets_list[i].data_ptr<int32_t>();
            }

            // choose kernel based on problem size
            if (hidden_dim <= 256 && num_tensors <= 16) {
                // use optimized kernel with shared memory and warp cooperation
                dim3 opt_blocks(num_rows);
                dim3 opt_threads(min(num_tensors * 32, 1024));  // each tensor gets one warp
                
                concat_2D_jagged_tensors_backward_kernel_opt<scalar_t><<<opt_blocks, opt_threads, 0, stream>>>(
                    grad_jagged_tensor,
                    num_tensors,
                    num_rows,
                    hidden_dim,
                    grad_output.data_ptr<scalar_t>(),
                    merged_offsets.data_ptr<int>()
                );
            } else {
                // use basic kernel for large problems
                concat_2D_jagged_tensors_backward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
                    grad_jagged_tensor,
                    num_tensors,
                    num_rows,
                    hidden_dim,
                    grad_output.data_ptr<scalar_t>(),
                    merged_offsets.data_ptr<int>()
                );
            }
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        }
    );
    return grad_inputs;
}
